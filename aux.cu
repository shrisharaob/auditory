#ifndef _AUX_
#define _AUX_
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include "globalVars.h"
#include "devFunctionProtos.h"
/* NVIDIA provides no linker so have to include SOURCE FILES to keep files of managble size */
#include "cudaRandFuncs.cu"
#include "isynap.cu"
#include "rkdumbPretty.cu"
#include "GenSparseMat.cu"
#include "bgCurrent.cu"
#include "IFF.cu"
#endif
